
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <sstream>
#include <ctime>
using namespace std;

// CONFIGURATION
const char* DATA_FILE = "IFK-8_OdinasT_L3_dat2.csv";
const int CAR_COUNT = 30;
const int RESULT_SIZE = 13;
const int THREAD_COUNT = 7;

const int GRADE_A_THRESHOLD = 15;
const int GRADE_B_THRESHOLD = 25;
//--

struct Car
{
	char Make[11];
	int Year;
	float Mileage;
};

void readData(const char* path, Car *cars)
{
	int i = 0;
	ifstream stream(path);
	string line;
	
	while (getline(stream, line))
	{
		istringstream str(line);
		string token;
		getline(str, token, ',');
		string make = token;
		getline(str, token, ',');
		int year = stoi(token);
		getline(str, token, ',');
		float mileage = stof(token);
		struct Car c;
		strcpy(c.Make, make.c_str());
		c.Year = year;
		c.Mileage = mileage;
		cars[i++] = c;
	}
	
	stream.close();
}

int getYear()
{
	time_t now = time(0);
	tm *ltm = localtime(&now);
	return 1900 + ltm->tm_year;
}

__global__ void run_on_gpu(Car *cars, int *year, char *results, int *result_count)
{
    int thread_count = blockDim.x;
	int i = threadIdx.x;

	while(i < CAR_COUNT)
	{
		int age = *year - cars[i].Year + cars[i].Mileage / 20000;
		char grade;
		
		if(age <= GRADE_A_THRESHOLD)
			grade = 'A';
		else if(age <= GRADE_B_THRESHOLD)
			grade = 'B';
		else
			grade = 'C';
			
		if (grade != 'C')
		{
			int index = atomicAdd(result_count, 1);
			index *= RESULT_SIZE;
			for (int j = 0; cars[i].Make[j] != 0; j++, index++)
				results[index] = cars[i].Make[j];
			results[index] = '-';
			index++;
			results[index] = grade;
		}
		i += thread_count;
	}
}

void writeResults(const char *path, const char *results, int count)
{
	ofstream stream(path);
	for (int i = 0; i < count * RESULT_SIZE; i++)
		stream << results[i];
	stream.close();
}

int main()
{
	int year = getYear(); //current year
	struct Car cars[CAR_COUNT];
	int result_count = 0;
	
	Car *device_cars;
	int *device_year;
	char *device_results;
	int *device_result_count;
	
	readData(DATA_FILE, cars);

	hipMalloc(&device_cars, sizeof(cars));
	hipMalloc(&device_year, sizeof(int));
	hipMalloc(&device_results, sizeof(char) * RESULT_SIZE * CAR_COUNT);
	hipMalloc(&device_result_count, sizeof(int));
	
	//copy data to GPU memory
	hipMemcpy(device_cars, cars, sizeof(cars), hipMemcpyHostToDevice);
	hipMemcpy(device_year, &year, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(device_result_count, &result_count, sizeof(int), hipMemcpyHostToDevice);
	
	run_on_gpu<<< 1, THREAD_COUNT >>>(device_cars, device_year, device_results, device_result_count);
	hipDeviceSynchronize();

	//get results back from GPU
	char results[RESULT_SIZE*CAR_COUNT];
	hipMemcpy(results, device_results, sizeof(results), hipMemcpyDeviceToHost);
	hipMemcpy(&result_count, device_result_count, sizeof(int), hipMemcpyDeviceToHost);
	
	writeResults("IFK-8_OdinasT_L3_rez.txt", results, result_count);

	//release GPU memory
	hipFree(device_cars);
	hipFree(device_year);
	hipFree(device_results);
	hipFree(device_result_count);
	
}